
#include <hip/hip_runtime.h>
extern "C" 
__global__ void throughData(double* A, double* B, int* C, int size) {
    int block = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int index = block * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    if(index >= size) return;

    if( A[index] >= B[index] )
        C[index] = 0;
    else C[index] = 1;
}